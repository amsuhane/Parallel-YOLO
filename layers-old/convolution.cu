#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

/*** Definitions ***/
// Block width for CUDA kernels
#define BW 128
#define RANDOM_SEED -1

#ifdef USE_GFLAGS
    #include <gflags/gflags.h>

    #ifndef _WIN32
        #define gflags google
    #endif
#else
    // Constant versions of gflags
    #define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
    #define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
    #define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
    #define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
    #define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)
/********************************************************************************************************************/


/*
Error Descriptions : HIPDNN_STATUS_BAD_PARAM : Some data is not given right
                     HIPDNN_STATUS_NOT_SUPPORTED : Given combination of inputs (descriptors) does not work
*/

///

class Conv {
public:
	// alpha and beta are scaling constants for the operations, use these default values
    const float alpha = 1.0f;
    const float beta = 0.0f;

    /* Tensor Descriptors for our operation */
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnTensorDescriptor_t bias_descriptor;
    hipdnnFilterDescriptor_t kernel_descriptor; // descriptor for the weight parameter
    hipdnnConvolutionDescriptor_t convolution_descriptor; // descriptor for the operation
    hipdnnConvolutionFwdAlgo_t convolution_algorithm; // descriptor for the algorithm to use
    hipdnnHandle_t cudnn;
    hipblasHandle_t cublas;

    size_t workspace = 0, tmpsize = 0;
    void* d_workspace{nullptr};
    size_t m_workspaceSize;

    hipdnnConvolutionBwdFilterAlgo_t convbwfalgo; // used for computing gradient with respect to weight
    hipdnnConvolutionBwdDataAlgo_t convbwdalgo; // used for computing gradient with respect to input
    bool falgo, dalgo; // if falgo, we compute gradient with respect to filter weight parameter, if dalgo, we compute gradient with respect to input

    /*** These variables are on GPU ***/
    // weights of the kernel and bias
    float *param_kernel;
    float *param_bias;

    // placeholders for gradients of parameters
    float *grad_kernel;
    float *grad_bias;
    float *grad_data; // gradient with respect input of convolution, Note : INPUT

    /*** These variables are on CPU ***/
    std::vector<float> cpu_param_kernel;
    std::vector<float> cpu_param_bias;

    /*** Definition variables we would be using ***/
    int input_size;
    int output_size;
    int out_height;
    int out_width;
    int gpu_id;
    int in_channels, kernel_size, out_channels;

    Conv(int _in_channels, int _out_channels, int _kernel_size, int padding, int stride, hipdnnHandle_t _cudnn, hipblasHandle_t _cublas,
         int batch_size, int width, int height, bool use_backward_filter, bool use_backward_data, int gpu_id,
         hipdnnTensorDescriptor_t& _input_descriptor, hipdnnTensorDescriptor_t& _output_descriptor, bool init_io_desc) {
    	/*
		use_backward_filter : Whether to compute gradient with respect to filter weights
		use_backward_data : Whether to compute gradient with respect to input
		init_io_desc : If true, the input and output descriptors are initialized from scratch else they are used as `_input_descriptor` and `_output_descriptor` as passed to the function
    	*/
        // Assign Handles
        cudnn = _cudnn;
        cublas = _cublas;
        // Assign the GPU id to run on
        gpu_id = gpu_id;
        checkCudaErrors(hipSetDevice(gpu_id));

        // Assign dimension values
        in_channels = _in_channels;
        out_channels = _out_channels;
        kernel_size = _kernel_size;
        out_width = ((width - kernel_size + 2*padding)/stride) + 1;
        out_height = ((height - kernel_size + 2*padding)/stride) + 1;

        /*** Forward Propagation Descriptors ***/
        // Input Tensor
        checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
        checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, // Uses Tensor Descriptor
                                              /*format=*/HIPDNN_TENSOR_NHWC,
                                              /*dataType=*/HIPDNN_DATA_FLOAT,
                                              /*batch_size=*/batch_size,
                                              /*channels=*/in_channels,
                                              /*image_height=*/height,
                                              /*image_width=*/width));

        // Output Tensor
        checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
        checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, // Uses Tensor Descriptor
                                              /*format=*/HIPDNN_TENSOR_NHWC,
                                              /*dataType=*/HIPDNN_DATA_FLOAT,
                                              /*batch_size=*/batch_size,
                                              /*channels=*/out_channels,
                                              /*image_height=*/out_height,
                                              /*image_width=*/out_width));

        // Bias Tensor
        checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                              HIPDNN_TENSOR_NHWC,
                                              HIPDNN_DATA_FLOAT,
                                              1, out_channels,
                                              1, 1));


        // Kernel Tensor
        
        checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
        checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor, // Uses Kernel Descriptor
                                              /*dataType=*/HIPDNN_DATA_FLOAT,
                                              /*format=*/HIPDNN_TENSOR_NHWC,
                                              /*out_channels=*/out_channels,
                                              /*in_channels=*/in_channels,
                                              /*kernel_height=*/kernel_size,
                                              /*kernel_width=*/kernel_size));

        /*** Create Convolution Descriptors ***/
        
        checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                                   /*pad_height=*/padding,
                                                   /*pad_width=*/padding,
                                                   /*vertical_stride=*/stride,
                                                   /*horizontal_stride=*/stride,
                                                   /*dilation_height=*/1,
                                                   /*dilation_width=*/1,
                                                   /*mode=*/HIPDNN_CONVOLUTION, // HIPDNN_CROSS_CORRELATION,
                                                   /*computeType=*/HIPDNN_DATA_FLOAT));

        /*** Create Convolution Algorithm Descriptors ***/
        if(init_io_desc)
            checkCUDNN(
                hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    convolution_descriptor,
                                                    output_descriptor,
                                                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                    /*memoryLimitInBytes=*/0,
                                                    &convolution_algorithm));
        else
            checkCUDNN(
                hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                    _input_descriptor, // init with what was passed to the function
                                                    kernel_descriptor,
                                                    convolution_descriptor,
                                                    _output_descriptor,
                                                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                    /*memoryLimitInBytes=*/0,
                                                    &convolution_algorithm));


        // /*** Allocating Memory To Workspace for the operations ***/
        if(init_io_desc)
            checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                               input_descriptor,
                                                               kernel_descriptor,
                                                               convolution_descriptor,
                                                               output_descriptor,
                                                               convolution_algorithm,
                                                               &workspace));
        else
            checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                                   _input_descriptor, // init with what was passed to the function
                                                                   kernel_descriptor,
                                                                   convolution_descriptor,
                                                                   _output_descriptor,
                                                                   convolution_algorithm,
                                                                   &workspace));

        /*** Backward Propagation Descriptors ***/
        // set falgo and dalgo
        falgo = use_backward_filter;
        dalgo = use_backward_data;
        //

        // If backprop filter algorithm was requested
        if (falgo)
        {   
            if(init_io_desc) {
                checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
                    cudnn, input_descriptor, output_descriptor, convolution_descriptor, kernel_descriptor,
                    HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &convbwfalgo));

                checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                    cudnn, input_descriptor, output_descriptor, convolution_descriptor, kernel_descriptor, 
                    convbwfalgo, &tmpsize));
            }
            else {
                checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
                cudnn, _input_descriptor, _output_descriptor, convolution_descriptor, kernel_descriptor,
                HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &convbwfalgo));

                checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                    cudnn, _input_descriptor, _output_descriptor, convolution_descriptor, kernel_descriptor, 
                    convbwfalgo, &tmpsize));
            }
        }

        workspace = std::max(workspace, tmpsize);

        // // If backprop data algorithm was requested
        if (dalgo)
        {
            if(init_io_desc) {
                checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(
                    cudnn, kernel_descriptor, output_descriptor, convolution_descriptor, input_descriptor,
                    HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &convbwdalgo));

                checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
                    cudnn, kernel_descriptor, output_descriptor, convolution_descriptor, input_descriptor, 
                    convbwdalgo, &tmpsize));
            }
            else {
                checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(
                    cudnn, kernel_descriptor, _output_descriptor, convolution_descriptor, _input_descriptor,
                    HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &convbwdalgo));

                checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
                    cudnn, kernel_descriptor, _output_descriptor, convolution_descriptor, _input_descriptor, 
                    convbwdalgo, &tmpsize));
            }
        }

        workspace = std::max(workspace, tmpsize);

        hipMalloc(&d_workspace, workspace);
        m_workspaceSize = workspace;

        /*** Allocate memory to kernel and bias ***/
        checkCudaErrors(hipMalloc(&param_kernel, sizeof(float) * in_channels * kernel_size * kernel_size * out_channels));
        checkCudaErrors(hipMalloc(&param_bias, sizeof(float) * out_channels));
        checkCudaErrors(hipMalloc(&grad_kernel, sizeof(float) * in_channels * kernel_size * kernel_size * out_channels));
        checkCudaErrors(hipMalloc(&grad_bias, sizeof(float) * out_channels));
        // Gradient with respect to output has same shape as output
        checkCudaErrors(hipMalloc(&grad_data,   sizeof(float) * batch_size * out_height * out_width * out_channels));

        input_size = batch_size * height * width * in_channels;
        output_size = batch_size * out_height * out_width * out_channels;

        // Initialie CPU-parameter memory
        cpu_param_kernel = std::vector<float>(in_channels * kernel_size * kernel_size * out_channels, 0);
        cpu_param_bias = std::vector<float>(out_channels, 0);

        // Initialize Parameters on GPU
        // init_weights();
        init_test_weights();

        // Move Initialized Weights to GPU
        // checkCudaErrors(hipMemcpyAsync(param_kernel, &cpu_param_kernel[0],     sizeof(float) * cpu_param_kernel.size(),  hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyAsync(param_bias, &cpu_param_bias[0], sizeof(float) * cpu_param_bias.size(),  hipMemcpyHostToDevice));
    }

    void init_test_weights() {
    	// Allocate Kernel
		const float kernel_template[2][2] = {
		  {1, 3},
		  {2, 4}
		};

		float h_kernel[1][2][2][1];
	    for (int row = 0; row < 2; ++row)
	      for (int column = 0; column < 2; ++column)
	        h_kernel[0][row][column][0] = kernel_template[row][column];

		hipMemcpy(param_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
    }

    void init_weights() {
    	// Initialize Weights
        std::random_device rd;
        std::mt19937 gen(RANDOM_SEED < 0 ? rd() : static_cast<unsigned int>(RANDOM_SEED));

        // Xavier Initialization
        float wconv = sqrt(3.0f / (kernel_size * kernel_size * in_channels));
        std::uniform_real_distribution<> dconv(-wconv, wconv);
        for (auto&& iter : cpu_param_kernel)
            iter = static_cast<float>(dconv(gen));
    }

    void forward(float *d_input, float *d_output) {
        checkCudaErrors(hipSetDevice(gpu_id));
        checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       param_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       m_workspaceSize,
                                       &beta,
                                       output_descriptor,
                                       d_output));

        checkCUDNN(hipdnnAddTensor(cudnn, &alpha, bias_descriptor,
                                  param_bias, &alpha, output_descriptor, d_output));
    }

    void backward(float *data_grad_above, hipdnnTensorDescriptor_t tensor_below, float *data_below) {
    	/*
		X : Input
		Y : Output
		W,b : Convolution Parameters

		Y = WX + b
		Y : Output of given convolution
		
		This calculates dW, db, dX

		data_grad_above : dY
		tensor_below : Descriptor of X
		data_below : X
    	*/
        checkCudaErrors(hipSetDevice(gpu_id));
        checkCUDNN(hipdnnConvolutionBackwardBias(cudnn, &alpha, output_descriptor,
                                                data_grad_above, &beta, bias_descriptor, grad_bias)); // correct!

        if(falgo)
            checkCUDNN(hipdnnConvolutionBackwardFilter(cudnn, &alpha, tensor_below,
                                                      data_below, output_descriptor, data_grad_above, convolution_descriptor,
                                                      convbwfalgo, d_workspace, m_workspaceSize,
                                                      &beta, kernel_descriptor, grad_kernel)); // workspace ka dekhna, baaki correct hai!
        
        if(dalgo)
            checkCUDNN(hipdnnConvolutionBackwardData(cudnn, &alpha, kernel_descriptor,
                                                    param_kernel, output_descriptor, data_grad_above, convolution_descriptor,
                                                    convbwdalgo, d_workspace, m_workspaceSize,
                                                    &beta, tensor_below, grad_data));
    }

    void updateWeights(float learning_rate) {
        int ks = in_channels * kernel_size * kernel_size * out_channels;
        int bs = out_channels;
        checkCudaErrors(hipblasSaxpy(cublas, static_cast<int>(ks),
                                    &alpha, grad_kernel, 1, param_kernel, 1));
        checkCudaErrors(hipblasSaxpy(cublas, static_cast<int>(bs),
                                    &alpha, grad_bias, 1, param_bias, 1));
    }

    void save_params(const char* fileprefix) {

        // get full filenames from the file prefix provided
        std::string param_kernel_file = std::string(fileprefix) + ".bin";
        std::string param_bias_file = std::string(fileprefix) + ".bias.bin";
        
        // Writing the weights to the file
        FILE *fp = fopen(param_kernel_file.c_str(), "wb");
        if (!fp) {
            printf("FILE ERROR: Cannot open file %s\n", param_kernel_file.c_str());
            exit(2);
        }
        fwrite(&cpu_param_kernel[0], sizeof(float), in_channels * kernel_size * kernel_size * out_channels, fp);
        fclose(fp);
    
        // Write the bias to the file
        fp = fopen(param_bias_file.c_str(), "wb");
        if (!fp) {
            printf("FILE ERROR: Cannot open file %s\n", param_bias_file.c_str());
            exit(2);
        }
        fwrite(&cpu_param_bias[0], sizeof(float), out_channels, fp);
        fclose(fp);
    
    }

    bool load_params(const char* fileprefix) {
        
        // get full filenames from the file prefix provided
        std::string param_kernel_file = std::string(fileprefix) + ".bin";
        std::string param_bias_file = std::string(fileprefix) + ".bias.bin";
        
        // reading the weights from the file
        FILE *fp = fopen(param_kernel_file.c_str(), "rb");
        if (!fp) {
            printf("FILE ERROR: Cannot open file %s\n", param_kernel_file.c_str());
            return false;
        }
        fread(&cpu_param_kernel[0], sizeof(float), in_channels * kernel_size * kernel_size * out_channels, fp);
        fclose(fp);
    
        // reading the bias from the file
        fp = fopen(param_bias_file.c_str(), "rb");
        if (!fp) {
            printf("FILE ERROR: Cannot open file %s\n", param_bias_file.c_str());
            return false;
        }
        fread(&cpu_param_bias[0], sizeof(float), out_channels, fp);
        fclose(fp);
    
        return true;
    
    }

};

void test_forward() {
	int WIDTH = 4, HEIGHT = 5, BATCH_SIZE = 1, CHANNELS = 1;
	int GPU_ID = 0;
	checkCudaErrors(hipSetDevice(GPU_ID));
    float *data, *output;
    hipdnnHandle_t cudnn;
    hipblasHandle_t cublas;

    hipdnnTensorDescriptor_t d1, d2; // dummy descriptors
    hipdnnCreate(&cudnn);
    hipblasCreate(&cublas);
    Conv c(1, CHANNELS, 2, 1, 1, cudnn, cublas,
         BATCH_SIZE, WIDTH, HEIGHT, true, false, GPU_ID, d1, d2, true);
    hipMalloc(&data, sizeof(float) * c.input_size);
    hipMalloc(&output, sizeof(float) * c.output_size);

    float *cpu_data = (float *)malloc(sizeof(float) * c.input_size);
    for(int i = 0;i < c.input_size;i++) cpu_data[i] = 1.0;
    checkCudaErrors(hipMemcpyAsync(data, cpu_data, sizeof(float) * c.input_size,  hipMemcpyHostToDevice));
	
    c.forward(data, output);

    // Move from device to host
    float *out = (float *)malloc(sizeof(float) * c.output_size);
    // float out[BATCH_SIZE][c.out_height][c.out_width][c.out_channels];
    checkCudaErrors(hipMemcpy(out, output, sizeof(float) * c.output_size, hipMemcpyDeviceToHost));

    // for(int i = 0;i < c.out_height;i++) {
    // 	for(int j = 0;j < c.out_width;j++)
    // 		std::cout << out[0][i][j][0] << " ";
    // 	std::cout << std::endl;
    // }
    // std::cout << std::endl;
    for(int i = 0;i < c.output_size;i++) {
    	std::cout << out[i] << " ";
    }
    std::cout << std::endl;
    std::cout << c.output_size << std::endl;
}

void test() {
    test_forward();
}

int main() {
    test();
    return 0;
}
