#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

/*** Definitions ***/
// Block width for CUDA kernels
#define BW 128
#define RANDOM_SEED -1

#ifdef USE_GFLAGS
    #include <gflags/gflags.h>

    #ifndef _WIN32
        #define gflags google
    #endif
#else
    // Constant versions of gflags
    #define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
    #define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
    #define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
    #define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
    #define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)
/********************************************************************************************************************/


/*
Error Descriptions : HIPDNN_STATUS_BAD_PARAM : Some data is not given right
                     HIPDNN_STATUS_NOT_SUPPORTED : Given combination of inputs (descriptors) does not work
*/

///

class FullyConnectedLayer {
public:
	// alpha and beta are scaling constants for the operations, use these default values
    const float alpha = 1.0f;
    const float beta = 0.0f;

    /* Tensor Descriptors for our operation */
    hipdnnTensorDescriptor_t input_tensor;
    hipdnnTensorDescriptor_t output_tensor;

    hipblasHandle_t cublas;

    /*** These variables are on GPU ***/
    // weights and bias
    float *weights;
    float *bias;

    /*** These variables are on CPU ***/
    std::vector<float> cpu_weights;
    std::vector<float> cpu_bias;

    int gpu_id;

    FullyConnectedLayer(int input_size, int output_size, int batch_size, hipblasHandle_t _cublas, int gpu_id,
         hipdnnTensorDescriptor_t& _input_descriptor, hipdnnTensorDescriptor_t& _output_descriptor) {
        
        // Create tensor for input (output from the pooling layer)
        checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));
        
        // Create tensor for output
        checkCUDNN(hipdnnCreateTensorDescriptor(&output_tensor));

        // Set tensor description
        checkCUDNN(hipdnnSetTensor4dDescriptor(input_tensor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            batch_size, input_size, 1, 1));

        checkCUDNN(hipdnnSetTensor4dDescriptor(output_tensor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            batch_size, output_size, 1, 1));

    }

    void init_test_weights() {
    }

    void init_weights() {
    }

    void forward(float *input_data, float *output_data, float, float *onevec) {

        // Forward propagation using weights
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            output_size, batch_size, input_size,
            &alpha,
            weights, input_size,
            input_data, input_size,
            &beta,
            output_data, output_size));

        // Adding bias to output_data
        checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            output_size, batch_size, 1,
            &alpha,
            bias, output_size,
            onevec, 1,
            &alpha,
            output_data, output_size));

    }

    void backward(float *data_grad_above, float *data_below, float* onevec) {
    }

    void updateWeights(float learning_rate) {
`       
        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.pneurons.size()),
        &alpha, gfc1, 1, pfc1, 1));

        checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.pbias.size()),
        &alpha, gfc1bias, 1, pfc1bias, 1));
    
    }

};

void test_forward() {
}

void test() {
    test_forward();
}

int main() {
    test();
    return 0;
}
