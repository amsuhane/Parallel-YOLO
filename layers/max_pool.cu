#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>


#include <hipblas.h>
#include <cudnn.h>

/*** Definitions ***/
// Block width for CUDA kernels
#define BW 128
#define RANDOM_SEED -1

#ifdef USE_GFLAGS
    #include <gflags/gflags.h>

    #ifndef _WIN32
        #define gflags google
    #endif
#else
    // Constant versions of gflags
    #define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
    #define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
    #define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
    #define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
    #define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != CUDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << cudnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)
/********************************************************************************************************************/

class MaxPoolLayer{
public:
    float alpha = 1.0f, beta = 0.0f;
    int gpu_id;

    cudnnPoolingDescriptor_t poolDesc;
    cudnnTensorDescriptor_t input_descriptor, poolTensor;

    cudnnHandle_t cudnnHandle;

    MaxPoolLayer(int size, int stride, int padding, int batch_size, int conv_out_channel, int conv_out_height, int conv_out_width, int _gpu_id, cudnnHandle_t _cudnnHandle){

        // Assign Handles
        cudnnHandle=_cudnnHandle;
    
        // Assign the GPU id to run on
        gpu_id = _gpu_id;
        checkCudaErrors(hipSetDevice(gpu_id));

        /*** Forward Propagation Descriptors ***/
        
        // Input Tensor (it is the output tensor from the convolution layer)
        checkCUDNN(cudnnCreateTensorDescriptor(&input_descriptor));
        checkCUDNN(cudnnSetTensor4dDescriptor(input_descriptor,                 // Uses Tensor Descriptor
                                              CUDNN_TENSOR_NHWC,                //format
                                              CUDNN_DATA_FLOAT,                 //dataType
                                              batch_size,                       //batch_size
                                              conv_out_channel,                 //channels
                                              conv_out_height,                  //image_height
                                              conv_out_width));                 //image_width

        // Pooling Descriptor
        checkCUDNN(cudnnCreatePoolingDescriptor(&poolDesc));            
        checkCUDNN(cudnnSetPooling2dDescriptor(poolDesc,
                                               CUDNN_POOLING_MAX,
                                               CUDNN_PROPAGATE_NAN,
                                               size, size,
                                               padding, padding,
                                               stride, stride));
        // Output Tensor
        checkCUDNN(cudnnCreateTensorDescriptor(&poolTensor));
        checkCUDNN(cudnnSetTensor4dDescriptor(poolTensor,
                                            CUDNN_TENSOR_NCHW,
                                            CUDNN_DATA_FLOAT,
                                            batch_size, 
                                            conv_out_channel,
                                            conv_out_height / stride,
                                            conv_out_width / stride));

        
    }

    void forward(float* input_tensor, float* output_tensor){
        checkCudaErrors(hipSetDevice(gpu_id));
        checkCUDNN(cudnnPoolingForward(cudnnHandle,         //handle
                                       poolDesc,            //poolingDesc
                                       &alpha,              //alpha
                                       input_descriptor,    //xDesc
                                       input_tensor,        //x
                                       &beta,               //beta
                                       poolTensor,          //yDesc
                                       output_tensor));     //y    
    }

    void backward(float *data_grad_from_above_conv, float *data_grad_below, cudnnTensorDescriptor_t output_tensor_of_below_conv, float *data_grad_above_from_below_conv, float* data_grad_below_from_below_conv, float* data_below, float* input_tensor, float* output_tensor){
        checkCUDNN(cudnnPoolingBackward(cudnnHandle,        //handle
                                        poolDesc,           //poolingDesc
                                        &alpha,             //alpha
                                        poolTensor,         //yDesc
                                        output_tensor,               //y
                                        poolTensor,         //dyDesc
                                        data_grad_from_above_conv,    //dy 
                                        output_tensor_of_below_conv,       //xDesc
                                        input_tensor,         //x     (NOT SURE ABOUT THIS)
                                        &beta,                             //beta
                                        output_tensor_of_below_conv,         //dxDesc
                                        data_grad_below_from_below_conv));                              //dx
    }

    void update_weights(){
        // No weights
        return;
    }
};

void test_mpl(){
  // Take 5x5 image, use 3x3 stride
  int WIDTH = 4, HEIGHT = 4, BATCH_SIZE = 1, CHANNELS = 1, SIZE=2, STRIDE=2, PADDING=0;
    float *data, *output;
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    MaxPoolLayer mpl(SIZE, STRIDE, PADDING, BATCH_SIZE, CHANNELS, HEIGHT, WIDTH, 0, cudnn);
    
    float* input_matrix = (float *)malloc(sizeof(float)*HEIGHT*WIDTH);
    float* output_matrix = (float *)malloc(sizeof(float)*(HEIGHT/STRIDE)*(WIDTH/STRIDE));
    for(int i=0; i<HEIGHT*WIDTH; i++) input_matrix[i]=i;
    hipMalloc(&data, sizeof(float) * WIDTH*HEIGHT);
    hipMalloc(&output, sizeof(float) * WIDTH*HEIGHT);
    checkCudaErrors(hipMemcpyAsync(data, input_matrix, sizeof(float)*HEIGHT*WIDTH, hipMemcpyHostToDevice));
    
    std::cout << "Input Matrix:\n";
    for(int i=0; i<HEIGHT*WIDTH; i++){
    if(i%WIDTH==0) std::cout << "\n";
    std::cout << input_matrix[i] << "  ";
  }
  
  std::cout << "\n\nPerforming max pool Size=" << SIZE << "x" << SIZE << " Stride=(" <<  STRIDE << ", " << STRIDE << ")\n";
  mpl.forward(data, output);
  
  checkCudaErrors(hipMemcpy(output_matrix, output, sizeof(float)*(HEIGHT/STRIDE)*(WIDTH/STRIDE), hipMemcpyDeviceToHost));
  std::cout << "\nOutput Matrix:\n";
  for(int i=0; i<(HEIGHT/STRIDE)*(WIDTH/STRIDE); i++){
    if(i%(WIDTH/STRIDE)==0) std::cout << "\n";
    std::cout << output_matrix[i] << " ";
  }
  std::cout << "\n";
}

int main() {
  test_mpl();
  return 0;
}